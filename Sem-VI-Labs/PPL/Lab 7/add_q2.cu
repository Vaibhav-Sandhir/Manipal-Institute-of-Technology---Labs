#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sort(int* matrix, int* ans, int n, int m){
	int i = blockIdx.x;
	int j = threadIdx.x;
	int ind = (i * m) + j;
	int start_of_row = (i * m);
	int end_of_row = (i * m) + m;
	int pos = start_of_row;
	int num = matrix[ind];
	for(int k = start_of_row; k < end_of_row; k++){
		if(num < matrix[k] || (num == matrix[k] && k < ind)){
			pos++;
		}
	}
	ans[pos] = num;
	return;
}

int main(){
	int n;
	int m;
	printf("\nEnter number of rows and columns: ");
	scanf("%d %d", &n, &m);
	int size = (n * m * sizeof(int));
	int* matrix = (int*)malloc(size);
	int* ans = (int*)malloc(size);
	int* cudaMatrix;
	int* cudaAns;
	printf("\nEnter values of the matrix: ");
	for(int i = 0; i < n * m; i++){
		scanf("%d", &matrix[i]);
	}
	hipMalloc(&cudaMatrix, size);
	hipMalloc(&cudaAns, size);
	hipMemcpy(cudaMatrix, matrix, size, hipMemcpyHostToDevice);
	dim3 dimGrid(n, 1, 1);
	dim3 dimBlock(m, 1, 1);
	sort <<<dimGrid, dimBlock>>> (cudaMatrix, cudaAns, n, m);
	hipMemcpy(ans, cudaAns, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n * m; i++){
		printf("%d ", ans[i]);
	}
	return 0;
}
