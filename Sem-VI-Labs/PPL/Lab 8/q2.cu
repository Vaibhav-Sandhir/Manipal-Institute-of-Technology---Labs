#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void transform(char* str, char* ans, int n){
	int id = threadIdx.x;
	int size = n - id;
	int ind = ((n - size) * (size + 1 + n)) / 2;
	for(int i = 0; i < size; i++){
		ans[ind + i] = str[i];
	}
	return;
}

int main(){
	char* str = (char*)malloc(1024 * sizeof(char));
	printf("\nEnter string: ");
	scanf("%s", str);
	int n = strlen(str);
	str[n] = '\0';
	int m = (n * (n + 1)) / 2;
	char* ans = (char*)malloc(m * sizeof(char));
	char* cudaStr;
	char* cudaAns;
	hipMalloc(&cudaStr, n * sizeof(char));
	hipMalloc(&cudaAns, m * sizeof(char));
	hipMemcpy(cudaStr, str, n * sizeof(char), hipMemcpyHostToDevice);
	transform<<<1, n>>> (cudaStr, cudaAns, n);
	hipMemcpy(ans, cudaAns, m * sizeof(char), hipMemcpyDeviceToHost);
	printf("\n%s", ans);
	return 0;
}
