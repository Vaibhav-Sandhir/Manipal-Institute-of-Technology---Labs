#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void even(int* arr, int n){
	int id = threadIdx.x;
	if(id % 2 == 0 && id < n - 1){
		if(arr[id] >= arr[id + 1]){
			int temp = arr[id];
			arr[id] = arr[id + 1];
			arr[id + 1] = temp;
		}
	}
	return;
}

__global__ void odd(int* arr, int n){
	int id = threadIdx.x;
	if(id % 2 == 1 && id < n - 1){
		if(arr[id] >= arr[id + 1]){
			int temp = arr[id];
			arr[id] = arr[id + 1];
			arr[id + 1] = temp;
		}
	}
	return;
}

int main(){
	int n;
	printf("\nEnter size of vector: ");
	scanf("%d", &n);
	int size = n * sizeof(int);
	int* arr = (int*)malloc(size);
	printf("\nEnter elements of the vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr[i]);
	}
	int* cudaArr;
	hipMalloc(&cudaArr, size);
	hipMemcpy(cudaArr, arr, size, hipMemcpyHostToDevice);
	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(n, 1, 1);
	for(int i = 0; i <= n / 2; i++){
		even <<<dimGrid, dimBlock>>> (cudaArr, n);
		odd <<<dimGrid, dimBlock>>> (cudaArr, n);
	}
	hipMemcpy(arr, cudaArr, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%d ", arr[i]);
	}
	return 0;
}
