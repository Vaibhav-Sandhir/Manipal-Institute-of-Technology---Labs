#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void addVectors(int* a, int* b, int* c){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = a[i] + b[i];
}

int main(){
	int n;
	printf("\nEnter number of length of the vectors: ");
	scanf("%d", &n);

	
	int* a = (int*)malloc(sizeof(int) * n);
	int* b = (int*)malloc(sizeof(int) * n);
	int* c = (int*)malloc(sizeof(int) * n);
	
	printf("\nEnter values of 1st vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &a[i]);
	}
	printf("\nEnter values of 2nd vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &b[i]);
	}
	
	int* cudaA;
	int* cudaB;
	int* cudaC;
	
	hipMalloc(&cudaA, sizeof(int) * n);
	hipMalloc(&cudaB, sizeof(int) * n);
	hipMalloc(&cudaC, sizeof(int) * n);
	
	// Destination source size direction
	hipMemcpy(cudaA, a, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(int) * n, hipMemcpyHostToDevice);
	
	// Grid Size Block Size
	addVectors<<<n, 1>>> (cudaA, cudaB, cudaC);
	// Cpu reads this as execute function addVectors in a cuda Kernel with grid size n and block size 1 with these parameters
	
	hipMemcpy(c, cudaC, sizeof(int) * n, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < n; i++){
		printf("%d ", c[i]);
	}  
	
	return 0;
}
