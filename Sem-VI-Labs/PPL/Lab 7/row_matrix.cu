#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sort(int* cudaMatrix, int n, int m){
	int i = blockIdx.x;
	int j = threadIdx.x;
	int ind = (i * m) + j;
	cudaMatrix[ind] = cudaMatrix[ind] + 1;
	return;
}

int main(){
	int n;
	int m;
	printf("\nEnter number of and rows and columns: ");
	scanf("%d %d", &n, &m);
	int size = (n * m * sizeof(int));
	int* matrix = (int*)malloc(size);
	int* cudaMatrix;
	printf("\nEnter values of the matrix: ");
	for(int i = 0; i < n * m; i++){
		scanf("%d", &matrix[i]);
	}
	hipMalloc(&cudaMatrix, size);
	hipMemcpy(cudaMatrix, matrix, size, hipMemcpyHostToDevice);
	dim3 dimGrid(n, 1, 1);
	dim3 dimBlock(m, 1, 1);
	
	sort <<<dimGrid, dimBlock>>> (cudaMatrix, n, m);
	hipMemcpy(matrix, cudaMatrix, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n * m; i++){
		printf("%d ", matrix[i]);
	}
	return 0;
}
