#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void convolution1D(double* arr, double* mask, double* res, int m, int n){
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	
	double value = 0;
	int start_point = i - (m / 2);
	for(int j = 0; j < m; j++){
		int ind = start_point + j;
		if(ind >= 0 && ind < n){
			value += arr[ind] * mask[j];
		}
	}
	res[i] = value;
}

int main(){
	int n;
	int m;
	printf("\nEnter length of the array: ");
	scanf("%d", &n);
	printf("\nEnter length of the mask: ");
	scanf("%d", &m);
	
	double* arr = (double*)malloc(sizeof(double) * n);
	double* mask = (double*)malloc(sizeof(double) * m);
	double* res = (double*)malloc(sizeof(double) *  n);
	
	printf("\nEnter element of the array: ");
	for(int i = 0; i < n; i++){
		scanf("%lf", &arr[i]);
	}
	
	printf("\nEnter elements of the mask: ");
	for(int i = 0; i < m; i++){
		scanf("%lf", &mask[i]);
	}
	
	double* cudaArr;
	double* cudaMask;
	double* cudaRes;
	
	hipMalloc(&cudaArr, sizeof(double) * n);
	hipMalloc(&cudaMask, sizeof(double) * m);
	hipMalloc(&cudaRes, sizeof(double) * n);
	
	hipMemcpy(cudaArr, arr, sizeof(double) * n, hipMemcpyHostToDevice);
	hipMemcpy(cudaMask, mask, sizeof(double) * m, hipMemcpyHostToDevice);
	
	convolution1D<<<1, n>>> (cudaArr, cudaMask, cudaRes, m, n);
	
	hipMemcpy(res, cudaRes, sizeof(double) * n, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < n; i++){
		printf("%lf ", res[i]);
	}
	
	return 0;
}
