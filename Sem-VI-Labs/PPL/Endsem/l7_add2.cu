#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sort(int* matrix, int* ans){
	int start_row = blockIdx.x * blockDim.x;
	int end_row = start_row + blockDim.x;
	int pos = start_row;
	int ind = start_row + threadIdx.x;
	int num = matrix[ind];
	for(int i = start_row; i < end_row; i++){
		if(matrix[i] > num || (matrix[i] == num && ind > i)){
			pos++;
		}
	}
	ans[pos] = num;
	return;
}

int main(){
	int m;
	int n;
	printf("\nEnter how many rows and columns: ");
	scanf("%d %d", &n, &m);
	int size = m * n * sizeof(int);
	int* matrix = (int*)malloc(size);
	int* ans = (int*)malloc(size);
	printf("\nEnter elements of the matrix: ");
	for(int i = 0; i < m * n; i++){
		scanf("%d", &matrix[i]);	
	}
	int* cudaMatrix;
	int* cudaAns;
	hipMalloc(&cudaMatrix, size);
	hipMalloc(&cudaAns, size);
	hipMemcpy(cudaMatrix, matrix, size, hipMemcpyHostToDevice);
	sort <<<m, n>>> (cudaMatrix, cudaAns);
	hipMemcpy(ans, cudaAns, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < m * n; i++){
		printf("%d ", ans[i]);	
	}
}
