#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void sinOf(double* a, double* b){
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	b[i] = sinf(a[i]);
}

int main(){
	int n;
	printf("\nEnter length of 1D array: ");
	scanf("%d", &n);
	int size = sizeof(double) * n;
	
	double* a = (double*)malloc(size);
	double* b = (double*)malloc(size);
	
	printf("\nEnter angles in radians: ");
	for(int i = 0; i < n; i++){
		scanf("%lf", &a[i]);
	}
	
	double* cudaA;
	double* cudaB;
	
	hipMalloc(&cudaA, size);
	hipMalloc(&cudaB, size);
	
	hipMemcpy(cudaA, a, size, hipMemcpyHostToDevice);
	
	sinOf<<<1, n>>>(cudaA, cudaB);
	
	hipMemcpy(b, cudaB, size, hipMemcpyDeviceToHost);
	
	printf("\nSin of all the angles: ");
	for(int i = 0; i < n; i++){
		printf("%lf ", b[i]);
	}
	
	return;
}
