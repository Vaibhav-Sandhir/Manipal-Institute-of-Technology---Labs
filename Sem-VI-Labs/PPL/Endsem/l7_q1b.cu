#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add(int* arr1, int* arr2, int* arr3){
	int i = threadIdx.x;
	arr3[i] = arr1[i] + arr2[i];
	return;
}

int main(){
	int n;
	printf("\nEnter how many elemenst are there in the vector: ");
	scanf("%d", &n);
	int size = n * sizeof(int);
	int* arr1 = (int*)malloc(size);
	int* arr2 = (int*)malloc(size);
	int* arr3 = (int*)malloc(size);
	printf("\nEnter elements of the first vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr1[i]);
	}
	printf("\nEnter elements of the second vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr2[i]);
	}
	int* cudaArr1;
	int* cudaArr2;
	int* cudaArr3;
	hipMalloc(&cudaArr1, size);
	hipMalloc(&cudaArr2, size);
	hipMalloc(&cudaArr3, size);
	hipMemcpy(cudaArr1, arr1, size, hipMemcpyHostToDevice);
	hipMemcpy(cudaArr2, arr2, size, hipMemcpyHostToDevice);
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(n, 1, 1);
	add <<<gridDim, blockDim>>>(cudaArr1, cudaArr2, cudaArr3);
	hipMemcpy(arr3, cudaArr3, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%d ", arr3[i]);
	}
	return 0;
}
