#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define THREADS 3

__global__ void add(int* arr1, int* arr2, int* arr3){
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	arr3[i] = arr2[i] + arr1[i];
	return;
}

int main(){
	int n;
	printf("\nEnter how many elements: ");
	scanf("%d", &n);
	int size = n * sizeof(int);
	int* arr1 = (int*)malloc(size);
	int* arr2 = (int*)malloc(size);
	int* arr3 = (int*)malloc(size);
	printf("\nEnter elements of the first vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr1[i]);
	} 
	printf("\nEnter elements of the second vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr2[i]);
	}
	int* cudaArr1;
	int* cudaArr2;
	int* cudaArr3;
	hipMalloc(&cudaArr1, size);
	hipMalloc(&cudaArr2, size);
	hipMalloc(&cudaArr3, size);
	hipMemcpy(cudaArr1, arr1, size, hipMemcpyHostToDevice);
	hipMemcpy(cudaArr2, arr2, size, hipMemcpyHostToDevice);
	dim3 gridDim((n + THREADS - 1) / THREADS, 1, 1);
	dim3 blockDim(THREADS, 1, 1);
	add <<<gridDim, blockDim>>> (cudaArr1, cudaArr2, cudaArr3);
	hipMemcpy(arr3, cudaArr3, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%d ", arr3[i]);
	}
	return 0;
}
