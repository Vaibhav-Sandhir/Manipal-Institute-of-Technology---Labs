#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void convolve(int* arr, int* mask, int* ans, int m, int n){
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	int start = i - m / 2;
	int sum = 0;
	for(int j = 0; j < m; j++){
		int ind = start + j;
		if(ind >= 0 && ind < n){
			sum += arr[ind] * mask[j];
		}
	}
	ans[i] = sum;
	return;
}

int main(){
	int n;
	int m;
	printf("\nEnter width of the array: ");
	scanf("%d", &n);
	printf("\nEnter width of the mask(odd): ");
	scanf("%d", &m);
	int* arr = (int*)malloc(n * sizeof(int));
	int* mask = (int*)malloc(m * sizeof(int));
	int* ans = (int*)malloc(n * sizeof(int));
	printf("\nEnter elements of the array: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr[i]);
	}
	printf("\nEnter elements of the mask: ");
	for(int i = 0; i < m; i++){
		scanf("%d", &mask[i]);
	}
	int* cudaArr;
	int* cudaMask;
	int* cudaAns;
	hipMalloc(&cudaArr, n * sizeof(int));
	hipMalloc(&cudaMask, m * sizeof(int));
	hipMalloc(&cudaAns, n * sizeof(int));
	hipMemcpy(cudaArr, arr, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaMask, arr, m * sizeof(int), hipMemcpyHostToDevice);
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(n , 1, 1);
	convolve <<<gridDim, blockDim>>> (cudaArr, cudaMask, cudaAns, m, n);
	hipMemcpy(ans, cudaAns, n * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%d ", ans[i]);
	}
	return 0;
}
