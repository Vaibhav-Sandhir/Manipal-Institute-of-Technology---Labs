#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sinOf(float* arr, float* ans){
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	ans[i] = sinf(arr[i]);
	return;
}

int main(){
	int n;
	printf("\nEnter how many elements: ");
	scanf("%d", &n);
	int size = n * sizeof(float);
	float* arr = (float*)malloc(size);
	float* ans = (float*)malloc(size);
	printf("\nEnter angles in radians: ");
	for(int i = 0; i < n; i++){
		scanf("%f", &arr[i]);
	}
	float* cudaArr;
	float* cudaAns;
	hipMalloc(&cudaArr, size);
	hipMalloc(&cudaAns, size);
	hipMemcpy(cudaArr, arr, size, hipMemcpyHostToDevice);
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(n, 1, 1);
	sinOf <<<gridDim, blockDim>>> (cudaArr, cudaAns);
	hipMemcpy(ans, cudaAns, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%f ", ans[i]);
	}
	return 0;
}
