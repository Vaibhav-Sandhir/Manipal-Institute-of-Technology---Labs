// y = mx + b

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add(int* x, int* b, int* y, int m){
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	y[i] = m * x[i] + b[i];
	return;
}

int main(){
	int n;
	int m;
	printf("\nEnter number of elements of the vector: ");
	scanf("%d", &n);
	int size = n * sizeof(int);
	int* x = (int*)malloc(size);
	int* b = (int*)malloc(size);
	int* y = (int*)malloc(size);
	printf("\nEnter elements of the vector x: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &x[i]);
	}
	printf("\nEnter value of slope m: ");
	scanf("%d", &m);
	printf("\nEnter value of vecotr b: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &b[i]);
	}
	int* cudaX;
	int* cudaB;
	int* cudaY;
	hipMalloc(&cudaX, size);
	hipMalloc(&cudaB, size);
	hipMalloc(&cudaY, size);
	hipMemcpy(cudaX, x, size, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, size, hipMemcpyHostToDevice);
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(n, 1, 1);
	add <<<gridDim, blockDim>>> (cudaX, cudaB, cudaY, m);
	hipMemcpy(y, cudaY, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%d ", y[i]);
	}
	return 0;
}
