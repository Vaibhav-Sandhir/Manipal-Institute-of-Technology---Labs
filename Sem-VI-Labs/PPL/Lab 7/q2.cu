#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>

__global__ void addVectors(int* a, int* b, int* c){
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	c[i] = a[i] + b[i];
	return;
}

int main(){
	int n;
	printf("\nEnter array length bigger than 2 : ");
	scanf("%d", &n);
	int size = sizeof(int) * n;
	
	int* a = (int*)malloc(size);
	int* b = (int*)malloc(size);
	int* c = (int*)malloc(size);
	
	printf("\nEnter elements of the 1st vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &a[i]);
	}
	printf("\nEnter elements of the 2nd vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &b[i]);
	}
	
	int* cudaA;
	int* cudaB;
	int* cudaC;
	
	hipMalloc(&cudaA, size);
	hipMalloc(&cudaB, size);
	hipMalloc(&cudaC, size);
	
	hipMemcpy(cudaA, a, size, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, size, hipMemcpyHostToDevice);
	
	int block_size = ceil(n / 2.0);
	addVectors<<<block_size, 2>>> (cudaA, cudaB, cudaC);
	
	hipMemcpy(c, cudaC, size, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < n; i++){
		printf("%d ", c[i]);
	}
	
	return 0;
}
