// Each column of resultant matrix is computed by one thread
/* m1 = 2  3  1  (2 x 3) (n x m)
        4  5  7 */
        
/* m2 = 1  8  5  (3 x 3) (p x q)
        4  2  7
        9  6  3 */
        
// m3 = (2 x 3) (n x q)
// m1 and m2 will be represented in a row major format

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void matmul(int* m1, int* m2, int* m3, int n, int m, int q){
	int cid_m2 = threadIdx.x;
	int sum;
	for(int rid_m1 = 0; rid_m1 < n; rid_m1++){
		sum = 0;
		for(int k = 0; k < m; k++){
			sum += m1[rid_m1 * m + k] * m2[k * q + cid_m2];
		}
		m3[rid_m1 * q + cid_m2] = sum;
	}
	return;
}


int main(){
	int n = 2;
	int m = 3;
	int p = 3;
	int q = 3;
	int m1[] = {2, 3, 1, 4, 5, 7};
	int m2[] = {1, 8, 5, 4, 2, 7, 9, 6, 3};
	int* m3 = (int*)malloc(n * q * sizeof(int));
	int* cudaM1;
	int* cudaM2;
	int* cudaM3;
	hipMalloc(&cudaM1, n * m * sizeof(int));
	hipMalloc(&cudaM2, p * q * sizeof(int));
	hipMalloc(&cudaM3, n * q * sizeof(int));
	hipMemcpy(cudaM1, m1, n * m * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaM2, m2, p * q * sizeof(int), hipMemcpyHostToDevice);
	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(q, 1, 1);
	matmul <<<dimGrid, dimBlock>>> (cudaM1, cudaM2, cudaM3, n, m, q);
	hipMemcpy(m3, cudaM3, n * q * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i < n * p; i++){
		printf("%d ", m3[i]);
	}
	return 0;
}

        
                        
