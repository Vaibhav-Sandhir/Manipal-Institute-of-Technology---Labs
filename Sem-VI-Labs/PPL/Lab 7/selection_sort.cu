#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sort(int* arr, int* ans, int n){
	int id = threadIdx.x + (blockDim.x * blockIdx.x);
	int num = arr[id];
	int pos = 0;
	for(int i = 0; i < n; i++){
		if(num < arr[i] || (num == arr[i] && i < id)){
			pos++;
		}
	}
	ans[pos] = num;
	return;
}

int main(){
	int n;
	printf("\nEnter size of vector: ");
	scanf("%d", &n);
	int size = n * sizeof(int);
	int* arr = (int*)malloc(size);
	printf("\nEnter elements of the vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr[i]);
	}
	int* cudaArr;
	int* cudaAns;
	hipMalloc(&cudaArr, size);
	hipMalloc(&cudaAns, size);
	hipMemcpy(cudaArr, arr, size, hipMemcpyHostToDevice);
	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(n, 1, 1);
	sort <<<dimGrid, dimBlock>>> (cudaArr, cudaAns, n);
	hipMemcpy(arr, cudaAns, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%d ", arr[i]);
	}
	return 0;
}
