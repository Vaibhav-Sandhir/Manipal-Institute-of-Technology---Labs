#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void multiply(int alpha, int* x, int* y, int* ans, int n){
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(n < 32){
		ans[i] = (alpha * x[i]) + y[i];
	}
	return;	
}

int main(){
	int n;
	printf("\nEnter vector length: ");
	scanf("%d", &n);
	int size = n * sizeof(n);
	int* x = (int*)malloc(size);
	int* y = (int*)malloc(size);
	int* ans = (int*)malloc(size);
	int alpha;
	printf("\nEnter values of x: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &x[i]);
	}
	printf("\nEnter values of y: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &y[i]);
	}
	printf("\nEnter constant value: ");
	scanf("%d", &alpha);
	int* cudaX;
	int* cudaY;
	int* cudaAns;
	hipMalloc(&cudaX, size);
	hipMalloc(&cudaY, size);
	hipMalloc(&cudaAns, size);
	hipMemcpy(cudaX, x, size, hipMemcpyHostToDevice);
	hipMemcpy(cudaY, y, size, hipMemcpyHostToDevice);
	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(n, 1, 1);
	multiply <<<dimGrid, dimBlock>>> (alpha, cudaX, cudaY, cudaAns, n);
	hipMemcpy(ans, cudaAns, size, hipMemcpyDeviceToHost);
	printf("\nAns is : ");
	for(int i = 0; i < n; i++){
		printf("%d ", ans[i]);
	}
	return 0;
}
