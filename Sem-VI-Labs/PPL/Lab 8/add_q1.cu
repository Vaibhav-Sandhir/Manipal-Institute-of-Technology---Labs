#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void reverse(char* str, int* pos){
	int id = threadIdx.x;
	int ind = 2 * id;
	int l = pos[ind];
	int r = pos[ind + 1];
	char temp;
	while(l <= r){
		temp = str[l];
		str[l] = str[r];
		str[r] = temp;
		l++;
		r--;
	}
	return;
}

int main(){
	char* str = (char*)malloc(1024 * sizeof(char));
	printf("\nEnter a string: ");
	fgets(str, 1024, stdin);
	int n = strlen(str);
	str[n] = '\0';
	char* ans = (char*)malloc(n * sizeof(char));
	int* pos = (int*)malloc(1024* sizeof(int));
	pos[0] = 0;
	int j = 1;
	for(int i = 0; i < n; i++){
		if(str[i] == ' '){
			pos[j++] = i - 1;
			pos[j++] = i + 1;
		}
	}
	pos[j++] = n - 1;
	pos[j] = '\0';
	int threads = j / 2;
	char* cudaStr;
	int* hipPos;
	hipMalloc(&cudaStr, n * sizeof(char));
	hipMalloc(&hipPos, j * sizeof(int));
	hipMemcpy(cudaStr, str, n * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(hipPos, pos, j * sizeof(int), hipMemcpyHostToDevice);
	reverse<<<1, threads>>> (cudaStr, hipPos);
	hipMemcpy(ans, cudaStr, n * sizeof(char), hipMemcpyDeviceToHost);
	printf("\nReversed String is : %s", ans);
	return 0;
}
