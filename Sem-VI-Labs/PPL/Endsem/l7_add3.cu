#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void odd(int* arr, int n){
	int i = threadIdx.x;
	if(i % 2 != 0 && i < n - 1){
		if(arr[i] > arr[i + 1]){
			int temp = arr[i];
			arr[i] = arr[i + 1];
			arr[i + 1] = temp;
		}
	}
}

__global__ void even(int* arr, int n){
	int i = threadIdx.x;
	if(i % 2 == 0 && i < n - 1){
		if(arr[i] > arr[i + 1]){
			int temp = arr[i];
			arr[i] = arr[i + 1];
			arr[i + 1] = temp;
		}
	}
}

int main(){
	int n;
	printf("\nEnter how many elements: ");
	scanf("%d", &n);
	int size = n * sizeof(int);
	int* arr = (int*)malloc(size);
	printf("\nEnter elements of the array: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &arr[i]);
	}
	int* cudaArr;
	hipMalloc(&cudaArr, size);
	hipMemcpy(cudaArr, arr, size, hipMemcpyHostToDevice);
	for(int i = 0; i < n - 1; i++){
		odd <<<1, n>>> (cudaArr, n);
		even <<<1, n>>> (cudaArr, n);
	}
	hipMemcpy(arr, cudaArr, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n; i++){
		printf("%d ", arr[i]);
	}
	return 0;
}
