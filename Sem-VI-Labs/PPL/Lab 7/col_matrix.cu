#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add(int* matrix, int n, int m){
	int col = blockIdx.x;
	int k = threadIdx.x;
	int i = (col * n) + k;
	
	matrix[i] = matrix[i] + 1;
	return; 
}

int main(){
	int n;
	int m;
	printf("\nEnter rows and columns of the matrix: ");
	scanf("%d %d", &n, &m);
	int size = m * n * sizeof(int);
	int* matrix = (int*)malloc(size);
	printf("\nEnter matrix values column wise: ");
	for(int i = 0; i < m * n; i++){
		scanf("%d", &matrix[i]);
	}
	
	int* cudaMatrix;
	hipMalloc(&cudaMatrix, size);
	hipMemcpy(cudaMatrix, matrix, size, hipMemcpyHostToDevice);
	dim3 dimGrid(m, 1, 1);
	dim3 dimBlock(n ,1 ,1);
	add <<<dimGrid, dimBlock>>> (cudaMatrix, n, m);
	hipMemcpy(matrix, cudaMatrix, size, hipMemcpyDeviceToHost);
	for(int i = 0; i < n * m; i++){
		printf("%d ", matrix[i]);
	}
	return 0;
}
