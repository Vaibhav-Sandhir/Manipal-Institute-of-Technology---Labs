#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void addVectors(int* a, int* b, int* c){
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	return;
}

int main(){
	int n;
	printf("\nEnter length of the vectors : ");
	scanf("%d", &n);
	
	int* a = (int*)malloc(sizeof(int) * n);
	int* b = (int*)malloc(sizeof(int) * n);
	int* c = (int*)malloc(sizeof(int) * n);
	
	printf("\nEnter values for the 1st vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &a[i]);
	}
	printf("\nEnter values for the 2nd vector: ");
	for(int i = 0; i < n; i++){
		scanf("%d", &b[i]);
	}
	
	int* cudaA;
	int* cudaB;
	int* cudaC;
	
	hipMalloc(&cudaA, sizeof(int) * n);
	hipMalloc(&cudaB, sizeof(int) * n);
	hipMalloc(&cudaC, sizeof(int) * n);
	
	hipMemcpy(cudaA, a, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(int) * n, hipMemcpyHostToDevice);
	
	addVectors<<<1, n>>> (cudaA, cudaB, cudaC);
	
	hipMemcpy(c, cudaC, sizeof(int) * n, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < n; i++){
		printf("%d ", c[i]);
	}  
	
	return 0;
}
