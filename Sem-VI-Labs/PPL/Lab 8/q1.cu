#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


__global__ void countWord(char* str, char* target, int* pos, int* count, int m){
	int id = threadIdx.x;
	int ind = 2 * id;
	int start = pos[ind];
	int end = pos[ind + 1];
	if(end - start + 1 != m){
		return;
	}
	for(int i = start, j = 0; i <= end; i++, j++){
		if(str[i] != target[j]){
			return;
		}
	}
	atomicAdd(count, 1);
	return;
}

int main(){
	char* str = (char*)malloc(1024 * sizeof(char));
	char* target = (char*)malloc(1024 * sizeof(char));
	int* count = 0;
	int* pos = (int*)malloc(1024 * sizeof(int));
	printf("\nEnter string: ");
	fgets(str, 1024, stdin);
	printf("\nEnter word to be searched: ");
	fgets(target, 1024, stdin);
	int n = strlen(str);
	int m = strlen(target);
	str[n] = '\0';
	target[m] = '\0';
	pos[0] = 0;
	int j = 1;
	for(int i = 0; i < n; i++){
		if(str[i] == ' '){
			pos[j++] = i - 1;
			pos[j++] = i + 1;
		}
	}
	pos[j++] = n - 1;
	pos[j] = '\0';
	int threads = j / 2;
	char* cudaStr;
	char* cudaTarget;
	int* hipPos;
	int* cudaCount = 0;
	int* result = (int*)malloc(sizeof(int));
	hipMalloc(&cudaStr, n * sizeof(char));
	hipMalloc(&cudaTarget, m * sizeof(char));
	hipMalloc(&hipPos, j * sizeof(int));
	hipMalloc(&cudaCount, sizeof(int));
	hipMemcpy(cudaStr, str, n * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(cudaTarget, target, strlen(target) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(hipPos, pos, j * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaCount, count, sizeof(int), hipMemcpyHostToDevice);
	countWord<<<1, threads>>> (cudaStr, cudaTarget, hipPos, cudaCount, m - 1);
	hipMemcpy(result, cudaCount, sizeof(int), hipMemcpyDeviceToHost);
	printf("\n%s occured %d times", target, *result);
	return 0; 	
}
